#include <iostream>
#include <hip/hip_runtime_api.h>

int _ConvertSMVer2Cores(int major, int minor) {
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM)
    struct SMInfo {
        int sm;  // 0xMm (hex), M = SM Major version, and m = SM minor version
        int cores;
    };

    SMInfo gpuArchCoresPerSM[] = {
        {0x10, 8},  // Tesla Generation (SM 1.0) G80 class
        {0x11, 8},  // Tesla Generation (SM 1.1) G8x class
        {0x12, 8},  // Tesla Generation (SM 1.2) G9x class
        {0x13, 8},  // Tesla Generation (SM 1.3) GT200 class
        {0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
        {0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
        {0x30, 192},// Kepler Generation (SM 3.0) GK10x class
        {0x32, 192},// Kepler Generation (SM 3.2) GK20A class
        {0x35, 192},// Kepler Generation (SM 3.5) GK11x class
        {0x37, 192},// Kepler Generation (SM 3.7) GK21x class
        {0x50, 128},// Maxwell Generation (SM 5.0) GM10x class
        {0x52, 128},// Maxwell Generation (SM 5.2) GM20x class
        {0x53, 128},// Maxwell Generation (SM 5.3) GM20x class
        {0x60, 64}, // Pascal Generation (SM 6.0) GP100 class
        {0x61, 128},// Pascal Generation (SM 6.1) GP10x class
        {0x62, 128},// Pascal Generation (SM 6.2) GP10x class
        {0x70, 64}, // Volta Generation (SM 7.0) GV100 class
        {0x72, 64}, // Volta Generation (SM 7.2) GV10B class
        {0x75, 64}, // Turing Generation (SM 7.5) TU10x class
        {0x80, 64}, // Ampere Generation (SM 8.0) GA100 class
        {0x86, 64}, // Ampere Generation (SM 8.6) GA10x class
        // Add future architectures here
        {-1, -1}    // Indicates unknown SM
    };

    int index = 0;
    while (gpuArchCoresPerSM[index].sm != -1) {
        if (gpuArchCoresPerSM[index].sm == ((major << 4) + minor)) {
            return gpuArchCoresPerSM[index].cores;
        }
        index++;
    }

    return -1; // Invalid architecture
}

// Structure to store GPU information
struct GPUInfo {
    std::string name;
    int driverVersion;
    int runtimeVersion;
    int major;
    int minor;
    float totalGlobalMemMB;
    int multiProcessorCount;
    int cudaCores;
    float maxClockRateGHz;
    size_t totalConstMem;
    size_t sharedMemPerBlock;
    int maxThreadsPerMultiProcessor;
    int maxThreadsPerBlock;
    int maxThreadsDim[3];
    int maxGridSize[3];
};

// Function to get GPU information
void getGPUInfo(GPUInfo &gpuInfo) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA-compatible GPU found." << std::endl;
        return;
    }

    // Assuming you want information about the first GPU
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    // Store GPU information in the struct
    gpuInfo.name = deviceProp.name;
    hipDriverGetVersion(&gpuInfo.driverVersion);
    hipRuntimeGetVersion(&gpuInfo.runtimeVersion);
    gpuInfo.major = deviceProp.major;
    gpuInfo.minor = deviceProp.minor;
    gpuInfo.totalGlobalMemMB = static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f);
    gpuInfo.multiProcessorCount = deviceProp.multiProcessorCount;
    gpuInfo.cudaCores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
    gpuInfo.maxClockRateGHz = deviceProp.clockRate * 1e-6f;
    gpuInfo.totalConstMem = deviceProp.totalConstMem;
    gpuInfo.sharedMemPerBlock = deviceProp.sharedMemPerBlock;
    gpuInfo.maxThreadsPerMultiProcessor = deviceProp.maxThreadsPerMultiProcessor;
    gpuInfo.maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    gpuInfo.maxThreadsDim[0] = deviceProp.maxThreadsDim[0];
    gpuInfo.maxThreadsDim[1] = deviceProp.maxThreadsDim[1];
    gpuInfo.maxThreadsDim[2] = deviceProp.maxThreadsDim[2];
    gpuInfo.maxGridSize[0] = deviceProp.maxGridSize[0];
    gpuInfo.maxGridSize[1] = deviceProp.maxGridSize[1];
    gpuInfo.maxGridSize[2] = deviceProp.maxGridSize[2];
}

// Function to print GPU information
void printGPUInfo(const GPUInfo &gpuInfo) {
    std::cout << "\nDevice: \"" << gpuInfo.name << "\"" << std::endl;

    std::cout << "  CUDA Driver Version / Runtime Version          "
              << gpuInfo.driverVersion / 1000 << "." << (gpuInfo.driverVersion % 100) / 10
              << " / " << gpuInfo.runtimeVersion / 1000 << "."
              << (gpuInfo.runtimeVersion % 100) / 10 << std::endl;

    std::cout << "  CUDA Capability Major/Minor version number:    "
              << gpuInfo.major << "." << gpuInfo.minor << std::endl;

    std::cout << "  Total amount of global memory:                 "
              << gpuInfo.totalGlobalMemMB << " MBytes" << std::endl;

    std::cout << "  ( " << gpuInfo.multiProcessorCount << ") Multiprocessors, ("
              << gpuInfo.multiProcessorCount * gpuInfo.cudaCores
              << ") CUDA Cores/MP:     " << gpuInfo.cudaCores
              << " CUDA Cores" << std::endl;

    std::cout << "  GPU Max Clock rate:                            "
              << gpuInfo.maxClockRateGHz << " GHz" << std::endl;

    std::cout << "  Total amount of constant memory:               "
              << gpuInfo.totalConstMem << " bytes" << std::endl;

    std::cout << "  Total amount of shared memory per block:       "
              << gpuInfo.sharedMemPerBlock << " bytes" << std::endl;

    std::cout << "  Maximum number of threads per multiprocessor:  "
              << gpuInfo.maxThreadsPerMultiProcessor << std::endl;

    std::cout << "  Maximum number of threads per block:           "
              << gpuInfo.maxThreadsPerBlock << std::endl;

    std::cout << "  Max dimension size of a thread block (x,y,z): ("
              << gpuInfo.maxThreadsDim[0] << ", " << gpuInfo.maxThreadsDim[1] << ", "
              << gpuInfo.maxThreadsDim[2] << ")" << std::endl;

    std::cout << "  Max dimension size of a grid size    (x,y,z): ("
              << gpuInfo.maxGridSize[0] << ", " << gpuInfo.maxGridSize[1] << ", "
              << gpuInfo.maxGridSize[2] << ")" << std::endl;
}

int main() {
    GPUInfo gpuInfo;
    getGPUInfo(gpuInfo);
    printGPUInfo(gpuInfo);
    return 0;
}

