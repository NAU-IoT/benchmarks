#include "hip/hip_runtime.h"
#include "../lib/primes.cuh"
#include <ctime>
#include <iostream>
#include <random>
#include <vector>
#include <hiprand/hiprand_kernel.h>

__device__ uint32_t gpu_mod_mul(uint32_t a, uint32_t b, uint32_t m) {
    uint32_t res = 0;
    while (b > 0) {
        if (b & 1) {
            res = (res + a) % m;
        }
        a = (2 * a) % m;
        b >>= 1;
    }
    return res;
}

__device__ uint32_t gpu_mod_pow(uint32_t a, uint32_t b, uint32_t m) {
    uint32_t res = 1;

    a %= m;
    while (b > 0) {
        if (b & 1) {
            res = gpu_mod_mul(res, a, m);
        }
        a = gpu_mod_mul(a, a, m);
        b >>= 1;
    }
    return res;
}

__device__ bool gpu_witness(uint32_t n, uint32_t d, uint32_t a, uint32_t s) {
    uint32_t x = gpu_mod_pow(a, d, n);

    if (x == 1 || x == n - 1) {
        return false;
    }

    for (uint32_t r = 1; r < s; r++) {
        x = gpu_mod_mul(x, x, n);
        if (x == n - 1) {
            return false;
        }
    }
    return true;
}

__global__ void
miller_rabin_kernel(const uint32_t *input, bool *output, int iters) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t num = input[idx];
    uint32_t d = num - 1, s = 0;

    if (num < 2) {
        output[idx] = false;
        return;
    }
    if (num == 2 || num == 3) {
        output[idx] = true;
        return;
    }
    if (num % 2 == 0) {
        output[idx] = false;
        return;
    }

    while (d % 2 == 0) {
        d /= 2;
        s++;
    }

    hiprandState state;
    hiprand_init(clock64(), idx, 0, &state);

    for (int i = 0; i < iters; i++) {
        uint32_t a = hiprand(&state) % (num - 3) + 2;
        if (gpu_witness(num, d, a, s)) {
            output[idx] = false;
            return;
        }
    }

    output[idx] = true;
}

void run_gpu_miller_rabin(const uint32_t *input, 
                          bool *output, 
                          int iters, 
                          int threads, 
                          int blocks) {

    miller_rabin_kernel<<<blocks, threads>>>(input,
                                             output,
                                             iters);
}

