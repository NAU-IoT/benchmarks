#include "hip/hip_runtime.h"
#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>

__device__ uint32_t mod_mul(uint32_t a, uint32_t b, uint32_t m) {
    uint32_t res = 0;
    while (b > 0) {
        if (b & 1) {
            res = (res + a) % m;
        }
        a = (2 * a) % m;
        b >>= 1;
    }
    return res;
}

__device__ uint32_t mod_pow(uint32_t a, uint32_t b, uint32_t m) {
    uint32_t res = 1;

    a %= m;
    while (b > 0) {
        if (b & 1) {
            res = mod_mul(res, a, m);
        }
        a = mod_mul(a, a, m);
        b >>= 1;
    }
    return res;
}

__device__ bool witness(uint32_t n, uint32_t d, uint32_t a, uint32_t s) {
    uint32_t x = mod_pow(a, d, n);

    if (x == 1 || x == n - 1) {
        return false;
    }

    for (uint32_t r = 1; r < s; r++) {
        x = mod_mul(x, x, n);
        if (x == n - 1) {
            return false;
        }
    }
    return true;
}

__global__ void
miller_rabin_kernel(const uint32_t *input, bool *output, int iters) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t num = input[idx];
    uint32_t d = num - 1, s = 0;

    if (num < 2) {
        output[idx] = false;
        return;
    }
    if (num == 2 || num == 3) {
        output[idx] = true;
        return;
    }
    if (num % 2 == 0) {
        output[idx] = false;
        return;
    }

    while (d % 2 == 0) {
        d /= 2;
        s++;
    }

    for (int i = 0; i < iters; i++) {
        uint32_t a = rand() % (num - 3) + 2;
        if (witness(num, d, a, s)) {
            output[idx] = false;
            return;
        }
    }
    output[idx] = true;
}

int main() {
    std::vector<uint32_t> nums = {
        1000000007, // A large 32-bit integer PRIME
        2147483647, // The largest 32-bit signed integer PRIME
        97,         // A PRIME number
        123456789,  // Another large 32-bit integer
        19,         // A PRIME number
        42,         // Just a random number
        31,         // A PRIME number
        987654321,  // Yet another large 32-bit integer
        37,         // A PRIME number
        123,        // Just another number
        17,         // A PRIME number
        999999999,  // And another large 32-bit integer
        23,         // A PRIME number
        777777777,  // Large 32-bit integer
        13,         // A PRIME number
        234567890,  // Large 32-bit integer
        11,         // A PRIME number
        987654321,  // Repeating value for demonstration
        7,          // A PRIME number
        8675309,    // Another large 32-bit integer
        709,        // A PRIME number
        5381,       // A PRIME number
        52711,      // A PRIME number
        167449,     // A PRIME number
        648391,     // A PRIME number
        1128889,    // A PRIME number
        2269733,    // A PRIME number
        3042161,    // A PRIME number
        4535189,    // A PRIME number
        7474967,    // A PRIME number
        9737333,    // A PRIME number
        14161729,   // A PRIME number
        17624813,   // A PRIME number
        19734581,   // A PRIME number
        23391799,   // A PRIME number
        29499439,   // A PRIME number
        37139213    // A PRIME number
    };

    int n = nums.size();
    int iters = 10;

    // Allocate device memory for input and output arrays
    uint32_t *d_input;
    bool *d_output;
    hipMalloc((void **)&d_input, n * sizeof(uint32_t));
    hipMalloc((void **)&d_output, n * sizeof(bool));

    // Copy input data to device
    hipMemcpy(d_input,
               nums.data(),
               n * sizeof(uint32_t),
               hipMemcpyHostToDevice);

    // Launch a separate thread for each element in the array
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    miller_rabin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input,
                                                            d_output,
                                                            iters);

    // Copy the results back to the host
    bool *results = new bool[n];
    hipMemcpy(results, d_output, n * sizeof(bool), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    delete[] results;

    return 0;
}
